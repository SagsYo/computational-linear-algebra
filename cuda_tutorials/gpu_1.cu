#include "hip/hip_runtime.h"
#include "book.h"

#define N 10


__global__ void add(int *a, int *b, int *c) {
    // int tid = blockIdx.x; // handle the data at this index
    int tid = threadIdx.x + blockIdx.x*blockDim.x; // handle the data at this index
    printf("Block ID x=%d y=%d\n", blockIdx.x, blockIdx.y);
    printf("Block Dim x=%d y=%d\n", blockDim.x, blockDim.y);
    printf("Thread ID x=%d y=%d\n", threadIdx.x, threadIdx.y);
    printf("Grid Dim x=%d y=%d\n", gridDim.x, gridDim.y);

    if (tid < N){
        c[tid] = (a[tid] + b[tid])/2;
    }
}

__global__ void assign(int *a, int *b){
    for (int i = 0; i < N; i++){
        a[i] = i;
        b[i] = i * i;
    }
    
}

int main( void ){
    int a[N], b[N], c[N];
    int *dev_a, *dev_b, *dev_c;

    HANDLE_ERROR(hipMalloc((void**) &dev_a, N*(sizeof(int))));
    HANDLE_ERROR(hipMalloc((void**) &dev_b, N*(sizeof(int))));
    HANDLE_ERROR(hipMalloc((void**) &dev_c, N*(sizeof(int))));
    HANDLE_ERROR(hipMemset(dev_c, 0, N*sizeof(int)));


    // assign<<<1, 1>>>(dev_a, dev_b);

    for (int i = 0; i < N; i++){
        a[i] = i;
        b[i] = i * i;
    }
    
    HANDLE_ERROR(hipMemcpy(dev_a, a, N*sizeof(int), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(dev_b, b, N*sizeof(int), hipMemcpyHostToDevice));


    // dim3 grid(N,N);
    // add<<<grid, grid>>>( dev_a, dev_b, dev_c );

    add<<<N/2, 2>>>( dev_a, dev_b, dev_c );

    HANDLE_ERROR(hipMemcpy(c, dev_c, N*sizeof(int), hipMemcpyDeviceToHost));

    // display the results
    for (int i = 0; i < N; i++){
        printf( "%d + %d = %d\n", a[i], b[i], c[i] );
    }


    hipFree( dev_a );
    hipFree( dev_b );
    hipFree( dev_c );

    return 0;
}
